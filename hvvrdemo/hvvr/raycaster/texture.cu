#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2017-present, Facebook, Inc. and its affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include "cuda_decl.h"
#include "cuda_util.h"
#include "material.h"
#include "texture.h"
#include "texture_internal.h"
#include "vector_math.h"

#include <vector>


namespace hvvr {

struct CudaFormatDescriptor {
    uint32_t r = 0, g = 0, b = 0, a = 0;
    hipChannelFormatKind channelType = hipChannelFormatKindNone;
    hipTextureReadMode readMode = hipReadModeElementType;
    bool sRGB = false;
    uint32_t elementSize = 0;
    CudaFormatDescriptor() {}
    CudaFormatDescriptor(uint32_t r,
                         uint32_t g,
                         uint32_t b,
                         uint32_t a,
                         hipChannelFormatKind channelType,
                         hipTextureReadMode readMode,
                         bool sRGB,
                         uint32_t elementSize)
        : r(r), g(g), b(b), a(a), channelType(channelType), readMode(readMode), sRGB(sRGB), elementSize(elementSize) {}
};

static CudaFormatDescriptor formatToDescriptor(TextureFormat format) {
    switch (format) {
        case TextureFormat::r8g8b8a8_unorm_srgb:
            return {8u, 8u, 8u, 8u, hipChannelFormatKindUnsigned, hipReadModeNormalizedFloat, true, 4};
        case TextureFormat::r8g8b8a8_unorm:
            return {8u, 8u, 8u, 8u, hipChannelFormatKindUnsigned, hipReadModeNormalizedFloat, false, 4};
		case TextureFormat::r16g16b16a16_unorm:
			return{ 16u, 16u, 16u, 16u, hipChannelFormatKindUnsigned, hipReadModeNormalizedFloat, false, 4 };
        case TextureFormat::r32g32b32a32_float:
            return {32u, 32u, 32u, 32u, hipChannelFormatKindFloat, hipReadModeElementType, false, 16};
        case TextureFormat::r16g16b16a16_float:
            return {16u, 16u, 16u, 16u, hipChannelFormatKindFloat, hipReadModeElementType, false, 8};
        case TextureFormat::r11g11b10_float:
            return {11u, 11u, 10u, 0u, hipChannelFormatKindFloat, hipReadModeElementType, false, 4};
        case TextureFormat::r32_float:
            return {32u, 0u, 0u, 0u, hipChannelFormatKindFloat, hipReadModeElementType, false, 4};
        default:
            printf("Unhandled texture format\n");
            assert(false);
    }
    return CudaFormatDescriptor();
}

Texture::Texture(const TextureData& textureData) {
    _textureID = CreateTexture(textureData);
}

// TODO(anankervis):
Texture::~Texture() {}


hipTextureObject_t* gDeviceTextureArray;
Texture2D gTextureAtlas[SimpleMaterial::maxTextureCount] = {};
static uint32_t gTextureCount = 0;

CUDA_DEVICE uchar4 to_uchar4(vector4 vec) {
    return make_uchar4((uint8_t)vec.x, (uint8_t)vec.y, (uint8_t)vec.z, (uint8_t)vec.w);
}

CUDA_KERNEL void d_mipmap(hipSurfaceObject_t mipOutput,
                          hipTextureObject_t mipInput,
                          uint32_t imageW,
                          uint32_t imageH) {
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    float px = 1.0 / float(imageW);
    float py = 1.0 / float(imageH);

    if ((x < imageW) && (y < imageH)) {
        // take the average of 4 samples

        // we are using the normalized access to make sure non-power-of-two textures
        // behave well when downsized.
        vector4 color = vector4(tex2D<float4>(mipInput, (x + 0) * px, (y + 0) * py)) +
                        vector4(tex2D<float4>(mipInput, (x + 1) * px, (y + 0) * py)) +
                        vector4(tex2D<float4>(mipInput, (x + 1) * px, (y + 1) * py)) +
                        vector4(tex2D<float4>(mipInput, (x + 0) * px, (y + 1) * py));

        color /= 4.0f;
        color *= 255.0f;
        color = min(color, 255.0f);

        surf2Dwrite(to_uchar4(color), mipOutput, x * sizeof(uchar4), y);
    }
}

static void generateMipMaps(hipMipmappedArray_t mipmapArray, uint32_t width, uint32_t height) {
#ifdef SHOW_MIPMAPS
    hipArray_t levelFirst;
    checkCudaErrors(hipGetMipmappedArrayLevel(&levelFirst, mipmapArray, 0));
#endif

    uint32_t level = 0;

    while (width != 1 || height != 1) {
        width /= 2;
        width = max(uint32_t(1), width);
        height /= 2;
        height = max(uint32_t(1), height);

        hipArray_t levelFrom;
        cutilSafeCall(hipGetMipmappedArrayLevel(&levelFrom, mipmapArray, level));
        hipArray_t levelTo;
        cutilSafeCall(hipGetMipmappedArrayLevel(&levelTo, mipmapArray, level + 1));

        hipExtent levelToSize;
        cutilSafeCall(hipArrayGetInfo(NULL, &levelToSize, NULL, levelTo));
        assert(levelToSize.width == width);
        assert(levelToSize.height == height);
        assert(levelToSize.depth == 0);

        // generate texture object for reading
        hipTextureObject_t texInput;
        hipResourceDesc texRes;
        memset(&texRes, 0, sizeof(hipResourceDesc));

        texRes.resType = hipResourceTypeArray;
        texRes.res.array.array = levelFrom;

        hipTextureDesc texDescr;
        memset(&texDescr, 0, sizeof(hipTextureDesc));

        texDescr.normalizedCoords = 1;
        texDescr.filterMode = hipFilterModeLinear;

        texDescr.addressMode[0] = hipAddressModeClamp;
        texDescr.addressMode[1] = hipAddressModeClamp;
        texDescr.addressMode[2] = hipAddressModeClamp;

        texDescr.readMode = hipReadModeNormalizedFloat;

        cutilSafeCall(hipCreateTextureObject(&texInput, &texRes, &texDescr, NULL));

        // generate surface object for writing

        hipSurfaceObject_t surfOutput;
        hipResourceDesc surfRes;
        memset(&surfRes, 0, sizeof(hipResourceDesc));
        surfRes.resType = hipResourceTypeArray;
        surfRes.res.array.array = levelTo;

        cutilSafeCall(hipCreateSurfaceObject(&surfOutput, &surfRes));

        // run mipmap kernel
        dim3 blockSize(16, 16, 1);
        dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y, 1);
        d_mipmap<<<gridSize, blockSize>>>(surfOutput, texInput, width, height);

        cutilSafeCall(hipDeviceSynchronize());
        cutilSafeCall(hipGetLastError());

        cutilSafeCall(hipDestroySurfaceObject(surfOutput));
        cutilSafeCall(hipDestroyTextureObject(texInput));

#ifdef SHOW_MIPMAPS
        // we blit the current mipmap back into first level
        hipMemcpy3DParms copyParams = {0};
        copyParams.dstArray = levelFirst;
        copyParams.srcArray = levelTo;
        copyParams.extent = make_hipExtent(width, height, 1);
        copyParams.kind = hipMemcpyDeviceToDevice;
        checkCudaErrors(hipMemcpy3D(&copyParams));
#endif

        level++;
    }
}

uint32_t getMipMapLevels(uint32_t width, uint32_t height, uint32_t depth) {
    uint32_t sz = max(max(width, height), depth);

    uint32_t levels = 0;
    while (sz) {
        sz /= 2;
        levels++;
    }

    return levels;
}

// CPU allocates resources address
uint32_t CreateTexture(const TextureData& textureData) {
    uint32_t depth = 0;

    assert(gTextureCount < SimpleMaterial::maxTextureCount - 1); // reserve the last index for SimpleMaterial::badTextureIndex

    if (gTextureCount == 0) {
        hipMalloc((void**)(&gDeviceTextureArray), sizeof(hipTextureObject_t) * SimpleMaterial::maxTextureCount);
    }

    CudaFormatDescriptor desc = formatToDescriptor(textureData.format);

    Texture2D tex;
    tex.width = textureData.width;
    tex.height = textureData.height;
    tex.elementSize = desc.elementSize;
    tex.hasMipMaps = true;
    tex.format = textureData.format;

    hipChannelFormatDesc chanDesc = hipCreateChannelDesc(desc.r, desc.g, desc.b, desc.a, desc.channelType);
    hipExtent extents = {textureData.width, textureData.height, depth};
    uint32_t levels = 0;
    if (tex.hasMipMaps) {
        // how many mipmaps we need
        levels = getMipMapLevels(textureData.width, textureData.height, depth);
        cutilSafeCall(hipMallocMipmappedArray(&tex.d_rawMipMappedMemory, &chanDesc, extents, levels));

        // upload level 0
        cutilSafeCall(hipGetMipmappedArrayLevel(&tex.d_rawMemory, tex.d_rawMipMappedMemory, 0));
    } else {
        // Create buffer for cuda write
        cutilSafeCall(hipMallocArray(&tex.d_rawMemory, &chanDesc, textureData.width, textureData.height));
    }

    hipTextureDesc texDesc = {};
    texDesc.addressMode[0] = hipAddressModeWrap;
    texDesc.addressMode[1] = hipAddressModeWrap;
    texDesc.addressMode[2] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = desc.readMode;
    texDesc.sRGB = desc.sRGB;
    texDesc.normalizedCoords = true;
    texDesc.maxAnisotropy = 8;

    printf("width: %u, height: %u, stride: %u, elementSize: %u\n", textureData.width, textureData.height,
           textureData.strideElements, desc.elementSize);
    cutilSafeCall(hipMemcpy2DToArray(tex.d_rawMemory, 0, 0, textureData.data, textureData.strideElements * desc.elementSize,
                                      textureData.width * desc.elementSize, textureData.height,
                                      hipMemcpyHostToDevice));

    hipResourceDesc resDesc = {};
    if (tex.hasMipMaps) {
        generateMipMaps(tex.d_rawMipMappedMemory, textureData.width, textureData.height);

        resDesc.resType = hipResourceTypeMipmappedArray;
        resDesc.res.mipmap.mipmap = tex.d_rawMipMappedMemory;

        texDesc.mipmapFilterMode = hipFilterModeLinear;
        texDesc.maxMipmapLevelClamp = float(levels - 1);
    } else {
        resDesc.resType = hipResourceTypeArray;
        resDesc.res.array.array = tex.d_rawMemory;
    }

    // Create Texture Object
    cutilSafeCall(hipCreateTextureObject(&tex.d_texObject, &resDesc, &texDesc, 0));
    cutilSafeCall(hipMemcpy(&gDeviceTextureArray[gTextureCount], &tex.d_texObject, sizeof(hipTextureObject_t),
                             hipMemcpyHostToDevice));

    gTextureAtlas[gTextureCount] = tex;
    ++gTextureCount;
    return gTextureCount - 1;
}

void DestroyAllTextures() {
    for (uint32_t i = 0; i < gTextureCount; ++i) {
        cutilSafeCall(hipFreeArray(gTextureAtlas[i].d_rawMemory));
        cutilSafeCall(hipDestroyTextureObject(gTextureAtlas[i].d_texObject));
    }
    cutilSafeCall(hipFree(gDeviceTextureArray));
    gTextureCount = 0;
}

Texture2D createEmptyTexture(uint32_t width,
                             uint32_t height,
                             TextureFormat format,
                             hipTextureAddressMode xWrapMode,
                             hipTextureAddressMode yWrapMode,
                             bool linearFilter) {
    CudaFormatDescriptor desc = formatToDescriptor(format);

    Texture2D tex;
    tex.width = width;
    tex.height = height;
    tex.elementSize = desc.elementSize;
    tex.hasMipMaps = false;
    tex.format = format;

    hipChannelFormatDesc chanDesc = hipCreateChannelDesc(desc.r, desc.g, desc.b, desc.a, desc.channelType);
    // Create buffer for cuda write
    cutilSafeCall(hipMallocArray(&tex.d_rawMemory, &chanDesc, width, height));

    hipTextureDesc texDesc = {};
    texDesc.addressMode[0] = xWrapMode;
    texDesc.addressMode[1] = yWrapMode;
    texDesc.addressMode[2] = hipAddressModeClamp;
    texDesc.filterMode = linearFilter ? hipFilterModeLinear : hipFilterModePoint;
    texDesc.readMode = desc.readMode;
    texDesc.normalizedCoords = true;
    texDesc.sRGB = desc.sRGB;

    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = tex.d_rawMemory;

    // Create Texture Object
    cutilSafeCall(hipCreateTextureObject(&tex.d_texObject, &resDesc, &texDesc, 0));
    // Create Surface Object
    cutilSafeCall(hipCreateSurfaceObject(&tex.d_surfaceObject, &resDesc));

    return tex;
}

CUDA_KERNEL void ClearKernel(Texture2D tex) {
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < tex.width*tex.elementSize && y < tex.height) {
		surf2Dwrite<unsigned char>(0, tex.d_surfaceObject, x, y);
	}
}

void clearTexture(Texture2D tex) {
	KernelDim dim(tex.width*tex.elementSize, tex.height, 16, 8);
	ClearKernel<<<dim.grid, dim.block>>>(tex);
}

} // namespace hvvr
