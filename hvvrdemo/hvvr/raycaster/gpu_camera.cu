/**
 * Copyright (c) 2017-present, Facebook, Inc. and its affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include "constants_math.h"
#include "gpu_camera.h"
#include "gpu_context.h"
#include "gpu_samples.cuh"
#include "kernel_constants.h"
#include "magic_constants.h"
#include "memory_helpers.h"
#ifdef _WIN32
#include <cuda_d3d11_interop.h>
#else
#include <GL/gl.h>
#include <cuda_gl_interop.h>
#endif
#include <functional>
#include <random>


namespace hvvr {

uint32_t pixelFormatSize(PixelFormat pixelFormat) {
    switch (pixelFormat) {
        case PixelFormat::RGBA8_SRGB:
            return 4;
        case PixelFormat::RGBA16:
            return 8;
        case PixelFormat::RGBA32F:
            return 16;
        default:
            assert(false);
            return 0;
    }
}

void Camera_StreamedData::reset(uint32_t tileCount) {
    tileCountOccupied = 0;
    tileCountEmpty = 0;

    tileIndexRemapEmpty.resizeDestructive(tileCount);
    tileIndexRemapOccupied.resizeDestructive(tileCount);
    tileTriRanges.resizeDestructive(tileCount);
    triIndices.resizeDestructive(MAX_TRI_INDICES_TO_INTERSECT);
    tileFrusta3D.resizeDestructive(tileCount);
}

GPUCamera::GPUCamera(const Camera* cameraPtr) : streamedIndexCPU(0), streamedIndexGPU(-1), cameraPtr(cameraPtr) {
    cutilSafeCall(hipStreamCreate(&stream));

    for (int n = 0; n < frameBuffering; n++) {
        cutilSafeCall(hipEventCreateWithFlags(&streamed[n].gpuDone, hipEventBlockingSync | hipEventDisableTiming));
    }
}
// TODO: there's no cleanup code for GPUCamera, yet, and it would be a big pain to clean it up to properly support
// the full set of constructors and assignments (especially move variants) given the number of members...

void GPUCamera::initLookupTables(int _MSAARate) {
    // getSubsampleUnitOffset needs a compile-time constant for MSAARate
    enum { MSAARate = COLOR_MODE_MSAA_RATE };
    if (MSAARate != _MSAARate)
        fail("MSAARate for lookup table must match compile-time constant\n");

    std::uniform_real_distribution<float> uniformRandomDist(0.0f, 1.0f);
    std::mt19937 generator;
    auto r = std::bind(uniformRandomDist, std::ref(generator));

    // lookup table for random lens position
    enum { TileCount = DOF_LENS_POS_LOOKUP_TABLE_TILES };
    std::vector<vector2> tileSubsampleLensPosData(TILE_SIZE * TileCount * MSAARate);
    for (int tile = 0; tile < TileCount; tile++) {
        for (int sample = 0; sample < int(TILE_SIZE); sample++) {
            float rotation = r() * Tau;

            for (int subsample = 0; subsample < MSAARate; subsample++) {
                vector2 pos =
                    getSubsampleUnitOffset<MSAARate>(vector2(0.0f, 0.0f), (subsample * 7 + 7) % MSAARate, rotation);

                // tileSubsampleLensPosData[tile * TILE_SIZE * MSAARate + subsample * TILE_SIZE + sample].x =
                //    uint32_t(floatToHalf(pos.x)) | (uint32_t(floatToHalf(pos.y)) << 16);
                tileSubsampleLensPosData[tile * TILE_SIZE * MSAARate + subsample * TILE_SIZE + sample] = pos;
            }
        }
    }
    d_tileSubsampleLensPos.resizeDestructive(TILE_SIZE * TileCount * MSAARate);
    d_tileSubsampleLensPos.upload(tileSubsampleLensPosData.data());
}

Camera_StreamedData* GPUCamera::streamedDataLock(uint32_t tileCount) {
    Camera_StreamedData* rval = streamed + streamedIndexCPU;
    cutilSafeCall(hipEventSynchronize(rval->gpuDone));
    streamedIndexCPU = (streamedIndexCPU + 1) % frameBuffering;
    rval->reset(tileCount);
    return rval;
}

void GPUCamera::streamedDataUnlock() {
    streamedIndexGPU = (streamedIndexGPU + 1) % frameBuffering;

    Camera_StreamedData* streamSrc = streamed + streamedIndexGPU;

    // some things don't have appropriate access patterns for reasonable PCIe streaming perf, so we copy them
    local.tileIndexRemapEmpty.resizeDestructive(streamSrc->tileIndexRemapEmpty.size());
    local.tileIndexRemapEmpty.uploadAsync(streamSrc->tileIndexRemapEmpty.data(), stream);

    local.tileIndexRemapOccupied.resizeDestructive(streamSrc->tileIndexRemapOccupied.size());
    local.tileIndexRemapOccupied.uploadAsync(streamSrc->tileIndexRemapOccupied.data(), stream);

    cutilFlush(stream);

    local.tileTriRanges.resizeDestructive(streamSrc->tileTriRanges.size());
    local.tileTriRanges.uploadAsync(streamSrc->tileTriRanges.data(), stream);

    local.tileFrusta3D.resizeDestructive(streamSrc->tileFrusta3D.size());
    local.tileFrusta3D.uploadAsync(streamSrc->tileFrusta3D.data(), stream);

    cutilFlush(stream);
}

void GPUCamera::streamedDataGpuDone() {
    cutilSafeCall(hipEventRecord(streamed[streamedIndexGPU].gpuDone, stream));
    cutilFlush(stream);
}

void GPUCamera::setCameraJitter(vector2 jitter) {
    frameJitter = jitter;
}

static int getMSAARate(RaycasterOutputFormat outputMode) {
    return (outputMode == RaycasterOutputFormat::COLOR_RGBA8) ? COLOR_MODE_MSAA_RATE : 1;
}

static TextureFormat pixelFormatToTextureFormat(PixelFormat format) {
    switch (format) {
        case PixelFormat::RGBA8_SRGB:
            return TextureFormat::r8g8b8a8_unorm_srgb;
        case PixelFormat::RGBA16:
            return TextureFormat::r16g16b16a16_unorm;
        case PixelFormat::RGBA32F:
            return TextureFormat::r32g32b32a32_float;
        default:
            assert(false);
    }
    return TextureFormat::none;
}

// TODO(anankervis): merge the different functions that duplicate camera resource creation
void GPUCamera::updateConfig(RaycasterOutputFormat _outputMode,
                             int32_t* sampleRemap,
                             DirectionalBeam* directionalSamples,
                             ThinLens _lens,
                             uint32_t _sampleCount,
                             uint32_t imageWidth,
                             uint32_t imageHeight,
                             uint32_t imageStride,
                             uint32_t _splitColorSamples) {
    splitColorSamples = _splitColorSamples;
    // one sample per output pixel, one sample per pentile subpixel, or one sample per R,G,B channel
    assert(splitColorSamples == 1 || splitColorSamples == 2 || splitColorSamples == 3);

    validSampleCount = imageWidth * imageHeight * splitColorSamples;
    d_sampleRemap = GPUBuffer<int32_t>(sampleRemap, sampleRemap + validSampleCount);
    sampleCount = _sampleCount;
    d_batchSpaceBeams = GPUBuffer<DirectionalBeam>(directionalSamples, directionalSamples + sampleCount);

    outputMode = _outputMode;
    int msaaRate = getMSAARate(outputMode);
    d_gBuffer = GPUBuffer<RaycasterGBufferSubsample>(sampleCount * msaaRate);

    PixelFormat outputFormat = outputModeToPixelFormat(outputMode);
    TextureFormat textureFormat = pixelFormatToTextureFormat(outputFormat);

    auto createImageSizedTexture = [&]() {
        return createEmptyTexture(imageWidth, imageHeight, textureFormat, hipAddressModeClamp, hipAddressModeClamp);
    };

    previousResultTexture = createImageSizedTexture();
    resultTexture = createImageSizedTexture();
    contrastEnhancementSettings.enable = true;
    contrastEnhancementSettings.f_e = 1.0f;
    contrastEnhancementBuffers.horizontallyFiltered = createImageSizedTexture();
    contrastEnhancementBuffers.fullyFiltered = createImageSizedTexture();

    auto pixelFormat = outputModeToPixelFormat(outputMode);
    d_sampleResults =
        GPUBuffer<uint32_t>((sampleCount * pixelFormatSize(pixelFormat) + sizeof(uint32_t) - 1) / sizeof(uint32_t));
    resultImage.update(imageWidth, imageHeight, imageStride, pixelFormat);
    lens = _lens;

    initLookupTables(msaaRate);
}

void GPUCamera::registerPolarFoveatedSamples(const std::vector<vector2ui>& polarRemapToPixel,
                                             float _maxEccentricityRadians,
                                             const EccentricityMap& eMap,
                                             uint32_t samplesPerRing,
                                             uint32_t paddedSampleCount) {
    PixelFormat outputFormat = outputModeToPixelFormat(outputMode);
    sampleCount = paddedSampleCount;
    d_sampleResults = GPUBuffer<uint32_t>((paddedSampleCount * pixelFormatSize(outputFormat) + sizeof(uint32_t) - 1) /
                                          sizeof(uint32_t));

    // For temporal filtering
    d_tMaxBuffer = GPUBuffer<float>(paddedSampleCount);
    eccentricityMap = eMap;
    maxEccentricityRadians = _maxEccentricityRadians;

    int msaaRate = getMSAARate(outputMode);
    size_t totalSubsampleCount = paddedSampleCount * msaaRate;

    // Allow us to launch a complete tile
    d_gBuffer = GPUBuffer<RaycasterGBufferSubsample>(totalSubsampleCount);

    d_polarRemapToPixel = makeGPUBuffer(polarRemapToPixel);

    TextureFormat textureFormat = pixelFormatToTextureFormat(outputFormat);

    uint32_t ringCount = uint32_t(polarRemapToPixel.size() / samplesPerRing);
    auto createFoveatedImage = [&](TextureFormat format, bool linearFilter = true) {
        return createEmptyTexture(samplesPerRing, ringCount, format, hipAddressModeWrap, hipAddressModeClamp,
                                  linearFilter);
    };
    polarTextures.raw = createFoveatedImage(textureFormat);
    polarTextures.depth = createFoveatedImage(TextureFormat::r32_float, false);
    polarTextures.moment1 = createFoveatedImage(TextureFormat::r16g16b16a16_unorm);
    polarTextures.moment2 = createFoveatedImage(TextureFormat::r16g16b16a16_unorm);

    initLookupTables(msaaRate);
}

bool GPUCamera::bindTexture(GPUContext& gpuContext, ImageResourceDescriptor texture) {
    if (resultsResource) {
        gpuContext.interopUnmapResources();
        cutilSafeCall(hipGraphicsUnregisterResource(resultsResource));
        resultsResource = nullptr;
    }
    if (texture.memoryType == ImageResourceDescriptor::MemoryType::DX_TEXTURE) {
#if defined(_WIN32)
        // hipGraphicsRegisterFlagsNone is only valid flag as of 7/22/2016
        cutilSafeCall(cudaGraphicsD3D11RegisterResource(&resultsResource, (ID3D11Texture2D*)texture.data,
                                                        hipGraphicsRegisterFlagsNone));
#else
//        cutilSafeCall(hipGraphicsGLRegisterBuffer(&resultsResource, ));
#endif
    } else if (texture.memoryType == ImageResourceDescriptor::MemoryType::OPENGL_TEXTURE) {
        cutilSafeCall(hipGraphicsGLRegisterImage(&resultsResource, (GLuint)(uint64_t)texture.data, GL_TEXTURE_2D,
                                                  cudaGraphicsMapFlagsWriteDiscard));
    }

    return true;
}

void GPUCamera::copyImageToBoundTexture() {
    hipArray* cuArray;
    cutilSafeCall(hipGraphicsSubResourceGetMappedArray(&cuArray, resultsResource, 0, 0));
    size_t srcStride = resultImage.width() * resultImage.bytesPerPixel(); // tightly packed
    cutilSafeCall(hipMemcpy2DToArrayAsync(cuArray, 0, 0, resultImage.data(), srcStride, srcStride,
                                           resultImage.height(), hipMemcpyDeviceToDevice, stream));
}

void GPUCamera::copyImageToCPU(ImageResourceDescriptor cpuTarget) {
    assert(!cpuTarget.isHardwareRenderTarget());
    auto pixFormat = outputModeToPixelFormat(outputMode);
    resultImage.update(cpuTarget.width, cpuTarget.height, (uint32_t)cpuTarget.stride, pixFormat);
    cutilSafeCall(
        hipMemcpyAsync(cpuTarget.data, resultImage.data(), resultImage.sizeInMemory(), hipMemcpyDeviceToHost, 0));
}


void GPUCamera::intersectShadeResolve(GPUSceneState& sceneState, const matrix4x4& cameraToWorld) {
    Camera_StreamedData& streamedData = streamed[streamedIndexGPU];

    // prep the scene
    sceneState.update();
    cutilSafeCall(hipStreamWaitEvent(stream, sceneState.updateEvent, 0));

    // The intersect and resolve kernels assume every thread will map to a valid work item, with valid input and output
    // slots. Sample count should be padded to a minimum of CUDA_GROUP_SIZE. In practice, it is padded to BLOCK_SIZE.
    assert(sampleCount % CUDA_GROUP_SIZE == 0);

    if (streamedData.tileCountEmpty > 0) {
        clearEmpty();
    }

    CameraBeams cameraBeams(*this);
    if (streamedData.tileCountOccupied > 0) {
        intersect(sceneState, cameraBeams, cameraToWorld);
        shadeAndResolve(sceneState, cameraBeams, cameraToWorld);
    }

    streamedDataGpuDone();
}

} // namespace hvvr
