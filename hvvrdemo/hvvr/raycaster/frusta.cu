#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2017-present, Facebook, Inc. and its affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include "constants_math.h"
#include "cuda_decl.h"
#include "foveated.h"
#include "frusta.h"
#include "gpu_buffer.h"
#include "gpu_camera.h"
#include "gpu_samples.cuh"
#include "kernel_constants.h"
#include "memory_helpers.h"
#include "sort.h"
#include "traversal.h"
#include "vector_math.h"
#include "warp_ops.h"


namespace hvvr {

void ComputeEyeSpaceFrusta(const GPUBuffer<DirectionalBeam>& dirSamples,
                           GPUBuffer<SimpleRayFrustum>& tileFrusta,
                           GPUBuffer<SimpleRayFrustum>& blockFrusta) {
    DynamicArray<DirectionalBeam> samples = makeDynamicArray(dirSamples);
    DynamicArray<SimpleRayFrustum> tFrusta = makeDynamicArray(tileFrusta);
    DynamicArray<SimpleRayFrustum> bFrusta = makeDynamicArray(blockFrusta);

    const bool checkFrustaAccuracy = false;
    const bool printStats = false;

    auto generateFrusta = [&checkFrustaAccuracy, &printStats](DynamicArray<SimpleRayFrustum>& frusta, unsigned int frustaSampleCount,
                             const DynamicArray<DirectionalBeam>& samples, float slopFactor, int numOrientationsToTry) {
        auto toDir = [](const matrix3x3& rot, float u, float v) { return rot * normalize(vector3(u, v, 1.0f)); };
        for (int i = 0; i < frusta.size(); ++i) {
            int sBegin = i * frustaSampleCount;
            int sEnd = min((int)((i + 1) * frustaSampleCount), (int)samples.size());
            vector3 dominantDirection(0.0f);
            for (int s = sBegin; s < sEnd; ++s) {
                dominantDirection += samples[s].centerRay;
            }
            dominantDirection = normalize(dominantDirection);

            // Try several different orientations for the plane, pick the one that
            // gives the smallest bounding box in uv space
            matrix3x3 rot(matrix3x3::rotationFromZAxis(dominantDirection));
            float bestUVArea = INFINITY;
            matrix3x3 bestRot;
            vector2 bestMinUV = vector2(INFINITY);
            vector2 bestMaxUV = vector2(-INFINITY);
            for (int o = 0; o < numOrientationsToTry; ++o) {
                const float range = (Pi / 2.0f) * 0.8f;
                matrix3x3 currRot =
                    matrix3x3::axisAngle(vector3(0, 0, 1), (range * o / float(numOrientationsToTry)) - (range / 2.0f)) *
                    rot;
                matrix3x3 invCurrRot = invert(currRot);
                vector2 minUV = vector2(INFINITY);
                vector2 maxUV = vector2(-INFINITY);
                for (int s = sBegin; s < sEnd; ++s) {
                    vector3 v = invCurrRot * samples[s].centerRay;
                    vector2 uv = vector2(v.x / v.z, v.y / v.z);
                    // TODO: check math here
                    v = invCurrRot * (samples[s].du + samples[s].centerRay);
                    float uvRadius = length(uv - vector2(v.x / v.z, v.y / v.z));
                    v = invCurrRot * (samples[s].dv + samples[s].centerRay);
                    uvRadius = max(uvRadius, length(uv - vector2(v.x / v.z, v.y / v.z)));
                    // slop; TODO: is this necessary, or can we do something more principled?
                    uvRadius *= slopFactor;
                    minUV = min(minUV, uv - uvRadius);
                    maxUV = max(maxUV, uv + uvRadius);
                }
                float uvArea = (maxUV.x - minUV.x) * (maxUV.y - minUV.y);
                if (uvArea < bestUVArea) {
                    bestRot = currRot;
                    bestUVArea = uvArea;
                    bestMinUV = minUV;
                    bestMaxUV = maxUV;
                }
            }

            SimpleRayFrustum f;
            for (int o = 0; o < 4; ++o) {
                f.origins[o] = {0.0f, 0.0f, 0.0f};
            }
            f.directions[0] = toDir(bestRot, bestMinUV.x, bestMaxUV.y);
            f.directions[1] = toDir(bestRot, bestMaxUV.x, bestMaxUV.y);
            f.directions[2] = toDir(bestRot, bestMaxUV.x, bestMinUV.y);
            f.directions[3] = toDir(bestRot, bestMinUV.x, bestMinUV.y);

            if (printStats) {
                for (int o = 0; o < 4; ++o) {
                    printf("f[%d].directions[%d]: %f, %f, %f\n", i, o, f.directions[o].x, f.directions[o].y,
                           f.directions[o].z);
                }
                printf("f[%d].bestUVArea: %f\n", i, bestUVArea);
                printf("Dominant Direction: %f %f %f\n", dominantDirection.x, dominantDirection.y, dominantDirection.z);
            }
            if (checkFrustaAccuracy) {
                // Make sure all samples points are within the frustum...
                Frustum checker(f.origins, f.directions);
                for (int s = sBegin; s < sEnd; ++s) {
                    auto C = samples[s].centerRay;
                    if (!frustumTestPoint(checker, C)) {
                        printf("TROUBLE: f[%d]: s[%d]:%f %f %f \n", i, s, C.x, C.y, C.z);
                    }
                }
            }

            frusta[i] = f;
        }
    };
    generateFrusta(tFrusta, TILE_SIZE, samples, 2.0f, 63);
    generateFrusta(bFrusta, BLOCK_SIZE, samples, 2.0f, 63);

    tileFrusta = makeGPUBuffer(tFrusta);
    blockFrusta = makeGPUBuffer(bFrusta);
}

} // namespace hvvr
