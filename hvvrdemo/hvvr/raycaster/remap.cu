#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2017-present, Facebook, Inc. and its affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include "gpu_camera.h"
#include "gpu_context.h"
#include "graphics_types.h"
#include "kernel_constants.h"
#include "shading_helpers.h"


namespace hvvr {

CUDA_DEVICE vector4 mergeSplitColors(const vector4* c) {
    return vector4(c[0].x, c[1].y, c[2].z, 1.0f);
}
CUDA_DEVICE uint32_t mergeSplitColors(const uint32_t* c) {
    return uint32_t((c[0] & 0xff) | (c[1] & 0xff00) | (c[2] & 0xff0000) | 0xff000000);
}

CUDA_DEVICE vector4 mergeSplitColorsPentile(const vector4* c, uint32_t x, uint32_t y) {
    return vector4(c[1].x, c[0].y, c[1].z, 1.0f);
}
CUDA_DEVICE uint32_t mergeSplitColorsPentile(const uint32_t* c, uint32_t x, uint32_t y) {
    uint32_t merged((c[1] & 0xff) | (c[0] & 0xff00) | (c[1] & 0xff0000) | 0xff000000);
    return merged;
}

// TODO(anankervis): for SplitColorSamples > 1, store each channel as a single component, instead of wasting space on
// RGBA PixelType
template <class PixelType, uint32_t SplitColorSamples>
CUDA_KERNEL void RemapKernel(PixelType* src,
                             int32_t* remap,
                             PixelType* dstImage,
                             uint32_t imageWidth,
                             uint32_t imageHeight,
                             uint32_t imageStride) {
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < imageWidth && y < imageHeight) {
        if (SplitColorSamples == 1) { // one sample per output pixel
            int32_t offset = remap[imageWidth * y + x];
            if (offset >= 0) {
                PixelType p = src[offset];
                dstImage[imageStride * y + x] = p;
            }
        } else if (SplitColorSamples == 2) { // one sample per channel, pentile
            PixelType splitColors[SplitColorSamples] = {};

            for (int channel = 0; channel < SplitColorSamples; channel++) {
                int32_t offset = remap[(imageWidth * y + x) * SplitColorSamples + channel];
                if (offset >= 0) {
                    splitColors[channel] = src[offset];
                }
            }

            PixelType outputColor = mergeSplitColorsPentile(splitColors, x, y);
            dstImage[imageStride * y + x] = outputColor;
        } else if (SplitColorSamples == 3) { // one sample per R,G,B channel
            PixelType splitColors[SplitColorSamples] = {};

            for (int channel = 0; channel < SplitColorSamples; channel++) {
                int32_t offset = remap[(imageWidth * y + x) * SplitColorSamples + channel];
                if (offset >= 0) {
                    splitColors[channel] = src[offset];
                }
            }

            PixelType outputColor = mergeSplitColors(splitColors);
            dstImage[imageStride * y + x] = outputColor;
        }
    }
}

void GPUCamera::remap() {

    KernelDim dim(resultImage.width(), resultImage.height(), CUDA_GROUP_WIDTH, CUDA_GROUP_HEIGHT);
    uint32_t* d_imageData = (uint32_t*)resultImage.data();
    switch (splitColorSamples) {
        case 1: {
            enum { SplitColorSamples = 1 };
            switch (outputModeToPixelFormat(outputMode)) {
                case PixelFormat::RGBA8_SRGB:
                    RemapKernel<uint32_t, SplitColorSamples><<<dim.grid, dim.block, 0, stream>>>(
                        d_sampleResults, d_sampleRemap, d_imageData, resultImage.width(), resultImage.height(),
                        resultImage.stride());
                    break;
                default:
                    assert(false);
            }
        } break;
        case 2: {
            enum { SplitColorSamples = 2 };
            switch (outputModeToPixelFormat(outputMode)) {
                case PixelFormat::RGBA8_SRGB:
                    RemapKernel<uint32_t, SplitColorSamples><<<dim.grid, dim.block, 0, stream>>>(
                        d_sampleResults, d_sampleRemap, d_imageData, resultImage.width(), resultImage.height(),
                        resultImage.stride());
                    break;
                default:
                    assert(false);
            }
        } break;
        case 3: {
            enum { SplitColorSamples = 3 };
            switch (outputModeToPixelFormat(outputMode)) {
                case PixelFormat::RGBA8_SRGB:
                    RemapKernel<uint32_t, SplitColorSamples><<<dim.grid, dim.block, 0, stream>>>(
                        d_sampleResults, d_sampleRemap, d_imageData, resultImage.width(), resultImage.height(),
                        resultImage.stride());
                    break;
                default:
                    assert(false);
            }
        } break;
        default:
            assert(false);
            break;
    }
}

// Switching to gather might improve performance, but this will likely never be a bottleneck
CUDA_KERNEL void RemapPolarFoveatedKernel(uint32_t* src,
                                          float* tmaxSrc,
                                          vector2ui* remap,
                                          Texture2D dstImage,
                                          Texture2D tmaxDstImage,
                                          size_t rawSampleCount) {
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < rawSampleCount) {
        vector2ui offset = remap[i];
        vector4 p = FromColor4Unorm8SRgb(src[i]);
        float t = tmaxSrc[i];
        surf2Dwrite(ToColor4Unorm8SRgb(p), dstImage.d_surfaceObject, offset.x * sizeof(uchar4), offset.y);
        surf2Dwrite(t, tmaxDstImage.d_surfaceObject, offset.x * sizeof(float), offset.y);
    }
}

void GPUCamera::remapPolarFoveated() {
    uint32_t rawSampleCount = polarTextures.raw.width * polarTextures.raw.height;
    KernelDim dim = KernelDim(rawSampleCount, CUDA_GROUP_SIZE);

    switch (outputModeToPixelFormat(outputMode)) {
        case PixelFormat::RGBA8_SRGB:
            RemapPolarFoveatedKernel<<<dim.grid, dim.block, 0, stream>>>(d_sampleResults.data(), d_tMaxBuffer,
                                                                         d_polarRemapToPixel, polarTextures.raw,
                                                                         polarTextures.depth, rawSampleCount);
            break;
        default:
            assert(false);
    }
}

} // namespace hvvr
